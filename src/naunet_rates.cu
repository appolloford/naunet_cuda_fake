#include "hip/hip_runtime.h"
#include <math.h>
/* */
#include <nvector/nvector_cuda.h>
#include <sunmatrix/sunmatrix_cusparse.h>
/* */
/*  */
#include "naunet_ode.h"
/*  */
#include "naunet_constants.h"
#include "naunet_macros.h"
#include "naunet_physics.h"

#define IJth(A, i, j) SM_ELEMENT_D(A, i, j)

// clang-format off
__device__ int EvalRates(realtype *k, realtype *y, NaunetData *u_data) {

    realtype nH = u_data->nH;
    realtype Tgas = u_data->Tgas;
    
    
    // clang-format on

    // Some variable definitions from krome
    realtype Te      = Tgas * 8.617343e-5;            // Tgas in eV (eV)
    realtype lnTe    = log(Te);                       // ln of Te (#)
    realtype T32     = Tgas * 0.0033333333333333335;  // Tgas/(300 K) (#)
    realtype invT    = 1.0 / Tgas;                    // inverse of T (1/K)
    realtype invTe   = 1.0 / Te;                      // inverse of T (1/eV)
    realtype sqrTgas = sqrt(Tgas);  // Tgas rootsquare (K**0.5)

    // reaaction rate (k) of each reaction
    // clang-format off
    k[0] = 3e-10;
        
    k[1] = 5e-11;
        
    
        // clang-format on

    return NAUNET_SUCCESS;
}